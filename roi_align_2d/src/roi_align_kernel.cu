#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include <stdio.h>
#include <math.h>
#include <float.h>
#include "roi_align_kernel.h"

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
            i += blockDim.x * gridDim.x)


  __global__ void ROIAlignForward(const int nthreads, const float* bottom_data, const float spatial_scale,
				  const int height, const int width, const int time, const int channels, const int aligned_height,
				  const int aligned_width, const int time_dim, const float* bottom_rois, float* top_data) {
        CUDA_1D_KERNEL_LOOP(index, nthreads) {
            // (n, c, t, ph, pw) is an element in the aligned output
            // int n = index;
            // int pw = n % aligned_width;
            // n /= aligned_width;
            // int ph = n % aligned_height;
            // n /= aligned_height;
            // int c = n % channels;
            // n /= channels;

            int pw = index % aligned_width;
            int ph = (index / aligned_width) % aligned_height;
	    int pt  = (index / aligned_width / aligned_height) % time_dim;
            int c  = (index / aligned_width / aligned_height / time_dim) % channels;
            int n  =  index / aligned_width / aligned_height / time_dim  / channels;

	    // if (index == 50 ){
	    //   printf("pw %d ph %d pt %d c %d n %d index %d \n",pw,ph,pt,c,n,index);
	    //   }

	    // get the rois

            float roi_batch_ind = bottom_rois[n * 7 + 0];
            float roi_start_w = bottom_rois[n * 7 + 1] * spatial_scale;
            float roi_start_h = bottom_rois[n * 7 + 2]  * spatial_scale;
	    float roi_start_t = bottom_rois[n * 7 + 3];
            float roi_end_w = bottom_rois[n * 7 + 4] * spatial_scale;
            float roi_end_h = bottom_rois[n * 7 + 5] * spatial_scale;
	    float roi_end_t = bottom_rois[n * 7 + 6];

	    // if(index == 50){
	    //   printf("0 n :%d roi_start_h : %f %f %f %f %f %f\n",n, float(roi_start_h),
	    // 	     float( roi_start_w), float( roi_start_t),
	    // 	     float( roi_end_w), roi_end_h,
	    // 	     float( roi_end_t));
	    // } 

            // // Force malformed ROIs to be 1x1
            float roi_width = fmaxf(roi_end_w - roi_start_w + 1., 0.);
            float roi_height = fmaxf(roi_end_h - roi_start_h + 1., 0.);
	    float roi_time = fmaxf(roi_end_t - roi_start_t + 1., 0.);
	    if (index == 50){
	      printf("roi_width = %f, roi_height %f, roi_time %f\n",roi_width,roi_height,roi_time);
	      printf("aligned_height %d aligned_width %d time_dim %d\n", aligned_height, aligned_width, time_dim);
	    }
            float bin_size_h = roi_height / (aligned_height - 1.);
            float bin_size_w = roi_width / (aligned_width - 1.);
	    float bin_size_t = roi_time / (time_dim - 1);

	    // if (index == 50){
	    //   printf("bin_size_h = %f, bin_size_w %f, bin_size_t %f\n",bin_size_h,bin_size_w,bin_size_t);
	    //   printf("aligned_height = %d, aligned_width %d, time_dim %d\n",aligned_height, aligned_width, time_dim);
	    // }

            float h = (float)(ph) * bin_size_h + roi_start_h;
            float w = (float)(pw) * bin_size_w + roi_start_w;
	    float t = (float)(pt) * bin_size_t + roi_start_t;

            int hstart = fminf(floor(h), height - 2);
            int wstart = fminf(floor(w), width - 2);
	    int tstart = fminf(floor(t), time - 2);

            int img_start = roi_batch_ind * channels * time * height * width;

	    // if (index == 50){
	    //   printf("roi_start_t %f\n",roi_start_t);
	    //   printf("pt :%d\n",pt);
	    //   printf("time %d t %f tstart %d\n",time, t, tstart);
	    // }
	    // if (index == 150){
	    //   printf("roi_start_t %f\n",roi_start_t);
	    //   printf("pt :%d\n",pt);
	    //   printf("time %d t %f tstart %d\n",time, t, tstart);
	    // }

            if (h < 0 || h >= height || w < 0 || w >= width || t < 0 || t >= time) {

                top_data[index] = 0.;

            } else {

                float h_ratio = h - (float)(hstart);
                float w_ratio = w - (float)(wstart);

                int upleft = img_start + ((c * time + tstart) *height + hstart) * width + wstart;
                int upright = upleft + 1;

                int downleft = upleft + width;
                int downright = downleft + 1;

        	top_data[index] = bottom_data[upleft] * (1. - h_ratio) * (1. - w_ratio)
		  + bottom_data[upright] * (1. - h_ratio) * w_ratio
		  + bottom_data[downleft] * h_ratio * (1. - w_ratio)
		  + bottom_data[downright] * h_ratio * w_ratio;
            }
        }
    }


  int ROIAlignForwardLaucher(const float* bottom_data, const float spatial_scale,  const int num_rois,
			     const int height, const int width, const int time, const int channels, const int aligned_height,
			     const int aligned_width, const int time_dim, const float* bottom_rois, float* top_data,
			     hipStream_t stream) {
        const int kThreadsPerBlock = 1024;
        const int output_size = num_rois * time_dim * aligned_height * aligned_width * channels;
        hipError_t err;


        ROIAlignForward<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>(
														  output_size, bottom_data, spatial_scale,  height,
														  width, time, channels, aligned_height, aligned_width,
														  time_dim, bottom_rois, top_data);

        err = hipGetLastError();
        if(hipSuccess != err) {
            fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
            exit( -1 );
        }
        return 1;
    }


  __global__ void ROIAlignBackward(const int nthreads, const float* top_diff, const float spatial_scale, 
				   const int height, const int width, const int time, const int channels, const int aligned_height,
				   const int aligned_width, const int time_dim, float* bottom_diff, const float* bottom_rois) {
        CUDA_1D_KERNEL_LOOP(index, nthreads) {

            // (n, c, ph, pw) is an element in the aligned output
            int pw = index % aligned_width;
            int ph = (index / aligned_width) % aligned_height;
	    int pt = (index / aligned_width / aligned_height) % time_dim;
            int c  = (index / aligned_width / aligned_height  / time_dim) % channels;
            int n  =  index / aligned_width / aligned_height  / time_dim  / channels;

            float roi_batch_ind = bottom_rois[n * 7 + 0];
            float roi_start_w = bottom_rois[n * 7 + 1] * spatial_scale;
            float roi_start_h = bottom_rois[n * 7 + 2] * spatial_scale;
	    float roi_start_t = bottom_rois[n * 7 + 3];
            float roi_end_w = bottom_rois[n * 7 + 4] * spatial_scale;
            float roi_end_h = bottom_rois[n * 7 + 5] * spatial_scale;
	    float roi_end_t = bottom_rois[n * 7 + 6];

            // Force malformed ROIs to be 1x1
            float roi_width = fmaxf(roi_end_w - roi_start_w + 1., 0.);
            float roi_height = fmaxf(roi_end_h - roi_start_h + 1., 0.);
	    float roi_time = fmaxf(roi_end_t - roi_start_t + 1., 0.);

            float bin_size_h = roi_height / (aligned_height - 1.);
            float bin_size_w = roi_width / (aligned_width - 1.);
	    float bin_size_t = roi_time / (time_dim - 1.);

            float h = (float)(ph) * bin_size_h + roi_start_h;
            float w = (float)(pw) * bin_size_w + roi_start_w;
	    float t = (float)(pt) * bin_size_t + roi_start_t;

            int hstart = fminf(floor(h), height - 2);
            int wstart = fminf(floor(w), width - 2);
	    int tstart = fminf(floor(t), time - 2);

            int img_start = roi_batch_ind * channels * time * height * width;

            // bilinear interpolation
            if (!(h < 0 || h >= height || w < 0 || w >= width || t < 0 || t >= time)) {

                float h_ratio = h - (float)(hstart);
                float w_ratio = w - (float)(wstart);

		// for the front bilinear interpolation

                int upleft = img_start + ((c * time + tstart) *height + hstart) * width + wstart;
                int upright = upleft + 1;

                int downleft = upleft + width;
                int downright = downleft + 1;

		// TODO understand what it does
                atomicAdd(bottom_diff + upleft, top_diff[index] * (1. - h_ratio) * (1 - w_ratio));
                atomicAdd(bottom_diff + upright, top_diff[index] * (1. - h_ratio) * w_ratio);
                atomicAdd(bottom_diff + downleft, top_diff[index] * h_ratio * (1 - w_ratio));
                atomicAdd(bottom_diff + downright, top_diff[index] * h_ratio * w_ratio);

            }
        }
    }

  int ROIAlignBackwardLaucher(const float* top_diff, const float spatial_scale, const int batch_size,
			      const int num_rois, const int height, const int width, const int time, const int channels,
			      const int aligned_height, const int aligned_width, const int time_dim, const float* bottom_rois,
			      float* bottom_diff, hipStream_t stream) {
        const int kThreadsPerBlock = 1024;
        const int output_size = num_rois * time_dim * aligned_height * aligned_width * channels;
        hipError_t err;

        ROIAlignBackward<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>(
														   output_size, top_diff, spatial_scale,
														   height, width, time, channels, aligned_height,
														   aligned_width, time_dim, bottom_diff, bottom_rois);

        err = hipGetLastError();
        if(hipSuccess != err) {
            fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
            exit( -1 );
        }

        return 1;
    }


#ifdef __cplusplus
}
#endif
