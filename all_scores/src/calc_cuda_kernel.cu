#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include <stdio.h>
#include <math.h>
#include <float.h>
#include "calc_cuda_kernel.h"

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
            i += blockDim.x * gridDim.x)


    __global__ void Calculate_scores(const int nthreads,const int K, const int N, const int array_size,
				    const float *actioness_scr, const float *overlaps_scr,
				    float *tube_scores) {
        CUDA_1D_KERNEL_LOOP(index, nthreads) {



	  int idx_i, idx_j, pre_idx_i, pre_idx_j;
	  int tmp_K;
	  float tmp_actioness_scr, tmp_overlaps_scr;

	  const int max_n_clips=5;
	  int pos[max_n_clips];
	  

	  tmp_actioness_scr = 0;
	  tmp_overlaps_scr  = 0;
	  
	  // if (index == 0){
	  //   printf("K %d, N %d, j %d z %d array_size %d\n", K,N,j,z, array_size);
	  // }

	  if ( index < array_size ) {

	    tmp_K = 1;

	    for(int i=N-1; i>=0; i--){
	      pos[i] = (index / tmp_K) % K;
	      tmp_K = tmp_K * K;
	    }

	    // if(index==134)
	    //   for (int i=0; i<N; i++)
	    // 	printf("i %d pos[i] %d\n",i, pos[i]);
		
	    for (int i=0; i<max_n_clips; i++){

	      if ( i >= max_n_clips )
		break;
	      
	      if ( i == 0 ){

		idx_i = 0;
		idx_j = pos[i];

		// if (index == 1){
		//   printf("mpike...\n");
		//   printf("idx_i :%d, idx_j :%d \n",idx_i, idx_j);
		//   printf("actioness_scr[idx_i*K+idx_j] %f \n", actioness_scr[idx_i*K+idx_j]);
		// }
		tmp_actioness_scr = actioness_scr[idx_i*K+idx_j];
	      }
	      else{

		pre_idx_i = idx_i;
		pre_idx_j = idx_j;

		idx_i = i;
		idx_j = pos[i];
		  
		// if (index == 0){
		//   printf("pre_idx_i :%d, pre_idx_j :%d \n",pre_idx_i, pre_idx_j);
		//   printf("idx_i :%d, idx_j :%d \n",idx_i, idx_j);
		//   printf("pred_idx_i*K*N*K :%d \n",pre_idx_i*K*K);
		//   printf("pre_idx_i*K*N*K + pre_idx_j*K :%d\n",pre_idx_i*K*K + pre_idx_j*K);
		//   printf("pre_idx_i*K*N*K + pre_idx_j*K + idx_j :%d\n",pre_idx_i*K*K + pre_idx_j*K+idx_j);
		// }

		tmp_actioness_scr += actioness_scr[idx_i*K+idx_j];
		tmp_overlaps_scr  += overlaps_scr[pre_idx_i*K*K + pre_idx_j*K + idx_j];

		// if (index == 1){
		//   printf("overlaps_scr[pre_idx_i*K*N*K + pre_idx_j*K + idx_j]; :%f \n",overlaps_scr[pre_idx_i*K*K + pre_idx_j*K + idx_j]);
		// }
	      }
	    }
	    if (N > 1){
	      // if (index == 0){
	      //   printf("tmp_actioness_scr :%f tmp_overlaps_scr %f\n", tmp_actioness_scr, tmp_overlaps_scr);
	      // }
		
	      tube_scores[index] = tmp_actioness_scr/N + tmp_overlaps_scr/(N-1);}
	    else
	      tube_scores[index] = tmp_actioness_scr;
	  }
	}
    }


    int CalculationLaucher(const int K, const int N, const int array_size, const float *actioness_scr,
			   const float *overlaps_scr, float *tube_scores, hipStream_t stream) {
        const int kThreadsPerBlock = 1024;
        // const int kThreadsPerBlock = 64;

        const int output_size = array_size * K;

        hipError_t err;
	// printf("output_size %d\n",output_size);
        Calculate_scores <<< (output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream >>>(
          output_size, K, N, array_size, actioness_scr, overlaps_scr, tube_scores);

        err = hipGetLastError();
        if(hipSuccess != err) {

            fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
            exit( -1 );
        }

        return 1;
    }


#ifdef __cplusplus
}
#endif
