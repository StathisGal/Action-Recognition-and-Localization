#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include <stdio.h>
#include <math.h>
#include <float.h>
#include "roi_align_kernel.h"

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
            i += blockDim.x * gridDim.x)


  __global__ void ROIAlignForward(const int nthreads, const float* bottom_data, const float spatial_scale, const float temp_scale,
				  const int height, const int width, const int time, const int channels, const int aligned_height,
				  const int aligned_width, const int time_dim, const float* bottom_rois, float* top_data) {
        CUDA_1D_KERNEL_LOOP(index, nthreads) {
            // (n, c, t, ph, pw) is an element in the aligned output
            // int n = index;
            // int pw = n % aligned_width;
            // n /= aligned_width;
            // int ph = n % aligned_height;
            // n /= aligned_height;
            // int c = n % channels;
            // n /= channels;

            int pw = index % aligned_width;
            int ph = (index / aligned_width) % aligned_height;
	    int pt  = (index / aligned_width / aligned_height) % time_dim;
            int c  = (index / aligned_width / aligned_height / time_dim) % channels;
            int n  =  index / aligned_width / aligned_height / time_dim  / channels;

            // bottom_rois += n * 5;
            float roi_batch_ind = bottom_rois[n * 7 + 0];
            float roi_start_w = bottom_rois[n * 7 + 1] * spatial_scale;
            float roi_start_h = bottom_rois[n * 7 + 2] * spatial_scale;
	    float roi_start_t = bottom_rois[n * 7 + 3] * temp_scale;
            float roi_end_w = bottom_rois[n * 7 + 4] * spatial_scale;
            float roi_end_h = bottom_rois[n * 7 + 5] * spatial_scale;
	    float roi_end_t = bottom_rois[n * 7 + 6] * temp_scale;

            // Force malformed ROIs to be 1x1
            float roi_width = fmaxf(roi_end_w - roi_start_w + 1., 0.);
            float roi_height = fmaxf(roi_end_h - roi_start_h + 1., 0.);
	    float roi_time = fmaxf(roi_end_t - roi_start_t + 1., 0.);

            float bin_size_h = roi_height / (aligned_height - 1.);
            float bin_size_w = roi_width / (aligned_width - 1.);
	    float bin_size_t = roi_time / (time_dim - 1.);

            float h = (float)(ph) * bin_size_h + roi_start_h;
            float w = (float)(pw) * bin_size_w + roi_start_w;
	    float t = (float)(pt) * bin_size_t + roi_start_t;

            int hstart = fminf(floor(h), height - 2);
            int wstart = fminf(floor(w), width - 2);
	    int tstart = fminf(floor(t), time - 2);

            int img_start = roi_batch_ind * channels * time * height * width;

            // trilinear interpolation = 2 bilinear interpolation + 1 linear interpolation
            if (h < 0 || h >= height || w < 0 || w >= width || t < 0 || w >= time) {
                top_data[index] = 0.;
            } else {
                float h_ratio = h - (float)(hstart);
                float w_ratio = w - (float)(wstart);
		float t_ratio = t - (float)(tstart);
		
		// for the front bilinear interpolation
                int upleftfront = img_start + ((c * time + tstart) *height + hstart) * width + wstart;
                int uprightfront = upleftfront + 1;

                int downleftfront = upleftfront + width;
                int downrightfront = downleftfront + 1;

		// for the back bilinear interpolation
                int upleftback = upleftfront + width * height;
                int uprightback = upleftback + 1;

                int downleftback = upleftback + width;
                int downrightback = downleftback + 1;

		float front_data = bottom_data[upleftfront] * (1. - h_ratio) * (1. - w_ratio)
                    + bottom_data[uprightfront] * (1. - h_ratio) * w_ratio
                    + bottom_data[downleftfront] * h_ratio * (1. - w_ratio)
                    + bottom_data[downrightfront] * h_ratio * w_ratio;

		float read_data = bottom_data[upleftback] * (1. - h_ratio) * (1. - w_ratio)
                    + bottom_data[uprightback] * (1. - h_ratio) * w_ratio
                    + bottom_data[downleftback] * h_ratio * (1. - w_ratio)
                    + bottom_data[downrightback] * h_ratio * w_ratio;

		top_data[index] = front_data * (1 - t_ratio) + read_data * t_ratio;

		  
            }
        }
    }


  int ROIAlignForwardLaucher(const float* bottom_data, const float spatial_scale, const float temp_scale, const int num_rois,
			     const int height, const int width, const int time, const int channels, const int aligned_height,
			     const int aligned_width, const int time_dim, const float* bottom_rois, float* top_data,
			     hipStream_t stream) {
        const int kThreadsPerBlock = 1024;
        const int output_size = num_rois * aligned_height * aligned_width * channels;
        hipError_t err;


        ROIAlignForward<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>(
														  output_size, bottom_data, spatial_scale, temp_scale, height,
														  width, time, channels, aligned_height, aligned_width,
														  time_dim, bottom_rois, top_data);

        err = hipGetLastError();
        if(hipSuccess != err) {
            fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
            exit( -1 );
        }

        return 1;
    }


  __global__ void ROIAlignBackward(const int nthreads, const float* top_diff, const float spatial_scale, const float temp_scale,
				   const int height, const int width, const int time, const int channels, const int aligned_height,
				   const int aligned_width, const int time_dim, float* bottom_diff, const float* bottom_rois) {
        CUDA_1D_KERNEL_LOOP(index, nthreads) {

            // (n, c, ph, pw) is an element in the aligned output
            int pw = index % aligned_width;
            int ph = (index / aligned_width) % aligned_height;
	    int pt = (index / aligned_width / aligned_height) % time_dim;
            int c  = (index / aligned_width / aligned_height  / time_dim) % channels;
            int n  =  index / aligned_width / aligned_height  / time_dim  / channels;

            float roi_batch_ind = bottom_rois[n * 7 + 0];
            float roi_start_w = bottom_rois[n * 7 + 1] * spatial_scale;
            float roi_start_h = bottom_rois[n * 7 + 2] * spatial_scale;
	    float roi_start_t = bottom_rois[n * 7 + 3] * temp_scale;
            float roi_end_w = bottom_rois[n * 7 + 4] * spatial_scale;
            float roi_end_h = bottom_rois[n * 7 + 5] * spatial_scale;
	    float roi_end_t = bottom_rois[n * 7 + 6] * temp_scale;
            /* int roi_start_w = round(bottom_rois[1] * spatial_scale); */
            /* int roi_start_h = round(bottom_rois[2] * spatial_scale); */
            /* int roi_end_w = round(bottom_rois[3] * spatial_scale); */
            /* int roi_end_h = round(bottom_rois[4] * spatial_scale); */

            // Force malformed ROIs to be 1x1
            float roi_width = fmaxf(roi_end_w - roi_start_w + 1., 0.);
            float roi_height = fmaxf(roi_end_h - roi_start_h + 1., 0.);
	    float roi_time = fmaxf(roi_end_t - roi_start_t + 1., 0.);
            float bin_size_h = roi_height / (aligned_height - 1.);
            float bin_size_w = roi_width / (aligned_width - 1.);
	    float bin_size_t = roi_width / (time_dim - 1.);

            float h = (float)(ph) * bin_size_h + roi_start_h;
            float w = (float)(pw) * bin_size_w + roi_start_w;
	    float t = (float)(pt) * bin_size_t + roi_start_t;

            int hstart = fminf(floor(h), height - 2);
            int wstart = fminf(floor(w), width - 2);
	    int tstart = fminf(floor(t), time - 2);

            int img_start = roi_batch_ind * channels * time * height * width;

            // bilinear interpolation
            if (!(h < 0 || h >= height || w < 0 || w >= width || t < 0 || t >= time)) {
                float h_ratio = h - (float)(hstart);
                float w_ratio = w - (float)(wstart);
		float t_ratio = t - (float)(tstart);

                // int upleft = img_start + (c * height + hstart) * width + wstart;
                // int upright = upleft + 1;
                // int downleft = upleft + width;
                // int downright = downleft + 1;

		// for the front bilinear interpolation

                int upleftfront = img_start + ((c * time + tstart) *height + hstart) * width + wstart;
                int uprightfront = upleftfront + 1;

                int downleftfront = upleftfront + width;
                int downrightfront = downleftfront + 1;

		// for the back bilinear interpolation
                int upleftback = upleftfront + width * height;
                int uprightback = upleftback + 1;

                int downleftback = upleftback + width;
                int downrightback = downleftback + 1;


		// TODO understand what it does
                atomicAdd(bottom_diff + upleftfront, top_diff[index] * (1. - h_ratio) * (1 - w_ratio) * (1- t_ratio));
                atomicAdd(bottom_diff + uprightfront, top_diff[index] * (1. - h_ratio) * w_ratio * (1- t_ratio));
                atomicAdd(bottom_diff + downleftfront, top_diff[index] * h_ratio * (1 - w_ratio) * (1- t_ratio));
                atomicAdd(bottom_diff + downrightfront, top_diff[index] * h_ratio * w_ratio *(1- t_ratio));

                atomicAdd(bottom_diff + upleftback, top_diff[index] * (1. - h_ratio) * (1 - w_ratio) * t_ratio);
                atomicAdd(bottom_diff + uprightback, top_diff[index] * (1. - h_ratio) * w_ratio * t_ratio);
                atomicAdd(bottom_diff + downleftback, top_diff[index] * h_ratio * (1 - w_ratio) * t_ratio);
                atomicAdd(bottom_diff + downrightback, top_diff[index] * h_ratio * w_ratio * t_ratio);
            }
        }
    }

  int ROIAlignBackwardLaucher(const float* top_diff, const float spatial_scale, const float temp_scale, const int batch_size,
			      const int num_rois, const int height, const int width, const int time, const int channels,
			      const int aligned_height, const int aligned_width, const int time_dim, const float* bottom_rois,
			      float* bottom_diff, hipStream_t stream) {
        const int kThreadsPerBlock = 1024;
        const int output_size = num_rois * aligned_height * aligned_width * channels;
        hipError_t err;

        ROIAlignBackward<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>(
														   output_size, top_diff, spatial_scale, temp_scale,
														   height, width, time, channels, aligned_height,
														   aligned_width, time_dim, bottom_diff, bottom_rois);

        err = hipGetLastError();
        if(hipSuccess != err) {
            fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
            exit( -1 );
        }

        return 1;
    }


#ifdef __cplusplus
}
#endif
